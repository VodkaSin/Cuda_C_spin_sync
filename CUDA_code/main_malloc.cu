#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#ifdef __unix__
#include <unistd.h>
#elif defined(_WIN32)|| defined(WIN32) 
#include <stdint.h>
#endif

#include "functions.cuh"
using namespace std;

#define PI (4.0 * atan(1.0));
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// Wrapper for if anything goes wrong with GPU
// e.g. gpuAssert(hipMalloc((void**)&para_a_dev,6*num_ens*sizeof(double)));
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
int loadDoubleData(char* filename, double* out);

/**
Arguments
POSITION	VARNAME		TYPE	NOTES
---------------------------------------------
argv[1]		num_ens: 	int		Number of classes
argv[2]		N_total: 	int		Total number of spins
argv[3]		theta_0: 	float	Coefficient of PI
argv[4]		phi_0: 		float	Coefficient of PI
argv[5]		coup_a_0:	float	Cavity-atom coupling strength
argv[6]		gamma_a_0:	float	Atom decay rate
argv[7]		chi_a_0:	float	Atom dephase rate
argv[8]		kappa_c_0:	float	Cavity decay rate
argv[9]		t_max:		float	Simulation end time
argv[10]	t_num:		int		Number of steps
argv[11]	handle: 	string	File handle to save

Example run:
file_alloc.exe 2 100000 1.0 0.0 1.6 0.0 0.0 160.0 2.0 60000 60000

To compile:
nvcc -w functions.cu main.cu -o file_alloc

To compile and run in one line
cls && nvcc -w functions.cu main_malloc.cu -o file_alloc && file_alloc.exe 2 100000 1.0 0.0 1.6 0.0 0.0 160.0 2.0 60000 60000

To run both file and file_alloc in one line
file.exe 128 100000 1.0  0.0 1.6 0.0 0.0 160.0 0.5 60000 ens_128 && file_alloc.exe 128 100000 1.0  0.0 1.6 0.0 0.0 160.0 0.5 60000 ens_128_alloc

*/
int main(int argc, char** argv) {

	// Print input values
	printf("num_ens:\t\t\t%s\n", argv[1]);
	printf("N_total:\t\t\t%s\n", argv[2]);
	printf("theta_0:\t\t\t%s\n", argv[3]);
	printf("phi_0:\t\t\t\t%s\n", argv[4]);
	printf("coup_a_0:\t\t\t%s\n", argv[5]);
	printf("gamma_a_0 (atom decay):\t\t%s\n", argv[6]);
	printf("chi_a_0 (atom dephase):\t\t%s\n", argv[7]);
	printf("kappa_c_0 (cavity decay):\t%s\n", argv[8]);
	printf("t_max:\t\t\t\t%s\n", argv[9]);
	printf("t_num:\t\t\t\t%s\n", argv[10]);
	printf("\n");

	//************************************************************************************** INITIAL PARAM *********************************
	// Ensemble settings
	int num_ens = atoi(argv[1]); 	// Number of classes
	int N_total = atoi(argv[2]); 	// Number of spins
	int ens_size = N_total/num_ens; // Number of spins in each class (uniform distribution)

	// Initial state
		// sin(theta_0/2)|e> + cos(theta_0/2)exp(i*phi_0)|g>
		// theta_0 = PI fully excited, theta_0 = 0 fully grounded
	double theta_0 = atof(argv[3])*PI;
	double phi_0 = atof(argv[4])*PI; 

	// System settings
		// Unit in kHz * 2pi
	double coup_a_0 =  atof(argv[5]); 	// Atom-cavity coupling
	double gamma_a_0 = atof(argv[6]); 	// Atom decay rate: [lower_a]
	double chi_a_0 =   atof(argv[7]); 	// Atom dephase rate: [sz]
	// SM Not taking effect yet
	double kappa_c_0 = atof(argv[8]); 	// Cavity decay rate: [a]
	double loss_0 =    0.0;				// Atom loss (population decreases rate)
	double omega_c =   0.0; 			// Cavity detuning
	double kappa_1_c = 1.0*100.0;		// LEFT MIRROR DECAY
	double kappa_2_c = 1.0*100.0;		// RIGHT MIRROR DECAY
	double eta_a_0 =   0.0;				// ATOM PUMPING

	//************************************************************************************** TIME CONSTANTS ********************************
	double t_max = atof(argv[9]);					// T_END
	int t_num = atoi(argv[10]);						// NUMBER OF STEPS
	double t_step = t_max/t_num;					// dT (SIZE OF EACH STEP)
	int t_store_num = 20000;
	int t_store =  t_num/t_store_num;
	
	// SGK check that t_num is larger than t_store_num, or it won't complete a run.
	if (t_num < t_store_num) {
		printf("[invalid param] Specify a 't_num' larger than or equal to %i", t_store_num);
		return;
	}

	// File handle
	char* handle = argv[11];

	// double inhomo[num_ens];
	double* inhomo = (double*)malloc(num_ens*sizeof(double));  // SGK

	// Writing in inhomo[]
	double maxdetun = 500;
	double sigma = 0.022;
	double sqrthalf = 0.707;
	// Deleted


	// Example of how to load detuning data into inhomo_test using `loadDoubleData()`
	// Define the number of rows and columns; for convenience
	int detuning_rows = 5;
	int detuning_cols = 2;
	// Initialize the array that we want to load the data into (use 1d array)
	double* inhomo_test = (double*)malloc(detuning_rows*detuning_cols*sizeof(double));
	// Load detuning data
	int res = loadDoubleData("Detuning.dat", inhomo_test);

	// Print out loaded data
	printf("Loaded detuning data:\n");
	for (int i=0; i<detuning_rows; i++) {
		for (int j=0; j<detuning_cols; j++) {
			printf("%f\t", inhomo_test[i*detuning_cols+j]);
		}
		printf("\n");
	}

	//********************************************************************************************* PARAMETERS FOR SQUARE PULSE ********************************************************

	double omega_d = 0.2;						// FREQUENCY OF SQUARE PULSE FOR INITIALIZATION
	//double coup_d =  0.0;					        // AMPLITUDE OF THE PULSE
	double coup_d = 0.0*3;					// AMPLITUDE OF THE PULSE
	double t_stop = 0.0*15; 					// LENGTH OF SQUARE PULSE in us
	//1.943*1.0E-7			


	//********************************************************************************************* PARAMETERS FOR OUTPUT POINTS *****************************************************

	// double N_a[num_ens],omega_a[num_ens],gamma_a[num_ens],\
			eta_a[num_ens],chi_a[num_ens],coup_a[num_ens],loss_a[num_ens];
	double* N_a = (double*)malloc(num_ens*sizeof(double));
	double* omega_a = (double*)malloc(num_ens*sizeof(double));
	double* gamma_a = (double*)malloc(num_ens*sizeof(double));
	double* eta_a = (double*)malloc(num_ens*sizeof(double));
	double* chi_a = (double*)malloc(num_ens*sizeof(double));
	double* coup_a = (double*)malloc(num_ens*sizeof(double));
	double* loss_a = (double*)malloc(num_ens*sizeof(double));
	// SGK

	for (int i =0; i < num_ens; i++){
		N_a[i] = ens_size;
		omega_a[i] = 10;
		gamma_a[i] = gamma_a_0;
		eta_a[i] = eta_a_0;
		chi_a[i] = chi_a_0;
		coup_a[i] = coup_a_0;
		loss_a[i] = loss_0;
	}


	// the parameters in an array 
	// double para_a[7*num_ens];
	// SGK
	double* para_a = (double*)malloc(7*num_ens*sizeof(double));

	for  (int i = 0; i < num_ens; i++){
		para_a[i] = N_a[i];
		para_a[i+num_ens] = omega_a[i];
		para_a[i+2*num_ens] = gamma_a[i];
		para_a[i+3*num_ens] = eta_a[i];
		para_a[i+4*num_ens] = chi_a[i];
		para_a[i+5*num_ens] = coup_a[i];
		para_a[i+6*num_ens] = loss_a[i];
	}

	// copy the parameters into the memory in GPU
	double *para_a_dev;
	hipMalloc((void**)&para_a_dev,6*num_ens*sizeof(double)); 
	hipMemcpy(para_a_dev,para_a,6*num_ens*sizeof(double),hipMemcpyHostToDevice);

	//*******************************
	// parameters for initial states 


	// double theta[num_ens],phi[num_ens];
	// SGK
	double* theta = (double*)malloc(num_ens*sizeof(double));
	double* phi = (double*)malloc(num_ens*sizeof(double));

	for (int i=0; i < num_ens; i++){
		theta[i] = theta_0;
		phi[i] = phi_0;
	}

	// double2 cu[num_ens],cl[num_ens];
	// SGK
	double2* cu = (double2*)malloc(num_ens*sizeof(double2));
	double2* cl = (double2*)malloc(num_ens*sizeof(double2));

	for (int i=0; i< num_ens; i++){
		cu[i].x = sin(0.5*theta[i])*cos(phi[i]);
		cu[i].y = sin(0.5*theta[i])*sin(phi[i]);
		
		cl[i].x = cos(0.5*theta[i]); 
		cl[i].y = 0.; 
	}

	double para_c[9];
	para_c[0] = omega_c;
	para_c[1] = kappa_1_c;
	para_c[2] = kappa_2_c;

	para_c[3] = omega_d;
	para_c[4] = coup_d;
	para_c[5] = t_stop;



	double *para_c_dev;
	hipMalloc((void**)&para_c_dev,9*sizeof(double));
	hipMemcpy(para_c_dev,para_c,9*sizeof(double),hipMemcpyHostToDevice);


	double *t_step_dev;
	hipMalloc((void**)&t_step_dev,sizeof(double));
	hipMemcpy(t_step_dev,&t_step,sizeof(double),hipMemcpyHostToDevice);







	// on CPU side 
	double2 ap_a,a,a_a;
	// double2 sz[num_ens],sm[num_ens],a_sz[num_ens],a_sm[num_ens],a_sp[num_ens];
	// double2 sm_sp[num_ens*num_ens],sm_sz[num_ens*num_ens],\
		sm_sm[num_ens*num_ens],sz_sz[num_ens*num_ens];

	double2* sz = (double2*)malloc(num_ens*sizeof(double2));
	double2* sm = (double2*)malloc(num_ens*sizeof(double2));
	double2* a_sz = (double2*)malloc(num_ens*sizeof(double2));
	double2* a_sm = (double2*)malloc(num_ens*sizeof(double2));
	double2* a_sp = (double2*)malloc(num_ens*sizeof(double2));
	double2* sm_sp = (double2*)malloc(num_ens*num_ens*sizeof(double2));
	double2* sm_sz = (double2*)malloc(num_ens*num_ens*sizeof(double2));
	double2* sm_sm = (double2*)malloc(num_ens*num_ens*sizeof(double2));
	double2* sz_sz = (double2*)malloc(num_ens*num_ens*sizeof(double2));

	// for initial values 
	double2 sm_1,sp_1,sz_1,sm_2,sz_2; 

	//****************************
	// initialize the observables
	ap_a.x = 0.; ap_a.y = 0.; a.x = 0.; a.y = 0.; a_a.x =0.; a_a.y = 0.; 

	for (int i= 0; i < num_ens; i++){
		sz_1.x = (cu[i].x*cu[i].x + cu[i].y*cu[i].y) - (cl[i].x*cl[i].x + cl[i].y*cl[i].y); 
		sz_1.y = 0.; 
		sm_1.x = cu[i].x*cl[i].x + cu[i].y*cl[i].y;
		sm_1.y = -cu[i].x*cl[i].y + cu[i].y*cl[i].x; 
		sp_1.x = cu[i].x*cl[i].x + cu[i].y*cl[i].y;
		sp_1.y = cu[i].x*cl[i].y - cu[i].y*cl[i].x;
		
		sz[i].x = sz_1.x; sz[i].y = sz_1.y;
		sm[i].x = sm_1.x; sm[i].y = sm_1.y; 
		
		a_sp[i].x = 0.; a_sp[i].y = 0.;
		a_sz[i].x = 0.; a_sz[i].y = 0.;
		a_sm[i].x = 0.; a_sm[i].y = 0.; 
		
		for (int j = 0; j < num_ens; j++){
			sz_2.x = (cu[j].x*cu[j].x + cu[j].y*cu[j].y) - (cl[j].x*cl[j].x + cl[j].y*cl[j].y); 
			sz_2.y = 0.; 
			sm_2.x = cu[j].x*cl[j].x + cu[j].y*cl[j].y;
			sm_2.y = -cu[j].x*cl[j].y + cu[j].y*cl[j].x; 
			
			sm_sp[j + i*num_ens].x = sm_2.x*sp_1.x - sm_2.y*sp_1.y; 
			sm_sp[j + i*num_ens].y = sm_2.x*sp_1.y + sm_2.y*sp_1.x; 
			
			sm_sz[j + i*num_ens].x = sm_2.x*sz_1.x - sm_2.y*sz_1.y;
			sm_sz[j + i*num_ens].y = sm_2.x*sz_1.y + sm_2.y*sz_1.x;
			
			sm_sm[j + i*num_ens].x = sm_2.x*sm_1.x - sm_2.y*sm_1.y;
			sm_sm[j + i*num_ens].y = sm_2.x*sm_1.y + sm_2.y*sm_1.x;
					
			sz_sz[j + i*num_ens].x = sz_2.x*sz_1.x - sz_2.y*sz_1.y;
			sz_sz[j + i*num_ens].y = sz_2.x*sz_1.y + sz_2.y*sz_1.x;	
		}
	}

	// on GUP side 
	double2 *ap_a_dev,*a_dev,*a_a_dev;
	double2 *sz_dev,*sm_dev,*a_sz_dev,*a_sm_dev,*a_sp_dev;
	double2 *sm_sp_dev,*sm_sz_dev,*sm_sm_dev,*sz_sz_dev;

	// create observables on GPU side 
	hipMalloc((void**)&ap_a_dev,sizeof(double2));
	hipMalloc((void**)&a_dev,sizeof(double2));
	hipMalloc((void**)&a_a_dev,sizeof(double2));

	hipMalloc((void**)&sz_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&sm_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&a_sz_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&a_sm_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&a_sp_dev,num_ens*sizeof(double2));

	hipMalloc((void**)&sm_sp_dev,num_ens*num_ens*sizeof(double2));
	hipMalloc((void**)&sm_sz_dev,num_ens*num_ens*sizeof(double2));
	hipMalloc((void**)&sm_sm_dev,num_ens*num_ens*sizeof(double2));
	hipMalloc((void**)&sz_sz_dev,num_ens*num_ens*sizeof(double2));



	// copy observables on GPU side 
	hipMemcpy(ap_a_dev,&ap_a,sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(a_dev,&a,sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(a_a_dev,&a_a,sizeof(double2),hipMemcpyHostToDevice);

	hipMemcpy(sz_dev,sz,num_ens*sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(sm_dev,sm,num_ens*sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(a_sz_dev,a_sz,num_ens*sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(a_sm_dev,a_sm,num_ens*sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(a_sp_dev,a_sp,num_ens*sizeof(double2),hipMemcpyHostToDevice);

	hipMemcpy(sm_sp_dev,sm_sp,num_ens*num_ens*sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(sm_sz_dev,sm_sz,num_ens*num_ens*sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(sm_sm_dev,sm_sm,num_ens*num_ens*sizeof(double2),hipMemcpyHostToDevice);
	hipMemcpy(sz_sz_dev,sz_sz,num_ens*num_ens*sizeof(double2),hipMemcpyHostToDevice);

	//***************
	// derivatives 
	double2 *d_ap_a_dev,*d_a_dev,*d_a_a_dev;
	double2 *d_sz_dev,*d_sm_dev,*d_a_sz_dev,*d_a_sm_dev,*d_a_sp_dev;
	double2 *d_sm_sp_dev,*d_sm_sz_dev,*d_sm_sm_dev,*d_sz_sz_dev;

	// create observables on GPU side 
	hipMalloc((void**)&d_ap_a_dev,sizeof(double2));
	hipMalloc((void**)&d_a_dev,sizeof(double2));
	hipMalloc((void**)&d_a_a_dev,sizeof(double2));

	hipMalloc((void**)&d_sz_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&d_sm_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&d_a_sz_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&d_a_sm_dev,num_ens*sizeof(double2));
	hipMalloc((void**)&d_a_sp_dev,num_ens*sizeof(double2));

	hipMalloc((void**)&d_sm_sp_dev,num_ens*num_ens*sizeof(double2));
	hipMalloc((void**)&d_sm_sz_dev,num_ens*num_ens*sizeof(double2));
	hipMalloc((void**)&d_sm_sm_dev,num_ens*num_ens*sizeof(double2));
	hipMalloc((void**)&d_sz_sz_dev,num_ens*num_ens*sizeof(double2));

	FILE *Result_time, *Result_Sz, *Result_photon, *Result_coherences_real;
	
	// time of simulation
	//************************************************************************************** OPEN FILE *********************************
		// Warning: handle must not be longer than 60 characters = keep it short
		// Space is allowed, just enclose with 
	char fname1[100];
	char fname2[100];
	char fname3[100];
	char fname4[100];


	snprintf(fname1, 100, "Result_time_%s.dat", handle);
	Result_time = fopen(fname1,"w");
	
	
	snprintf(fname2, 100, "Result_Sz_%s.dat", handle);
	Result_Sz = fopen(fname2,"w");

	snprintf(fname3, 100, "Result_photon_%s.dat", handle);
	Result_photon= fopen(fname3,"w");

	snprintf(fname4, 100, "Result_coherences_real_%s.dat", handle);
	Result_coherences_real= fopen(fname4,"w");


	// ***********************************
	// simulations starts
	// ***********************************
	clock_t start_clock, end_clock;
	start_clock = clock();
	double tc;


	// update the old reduced density matrix 
	for (int t = 1; t < t_num; t++){
		// printf("t %i of t_num %i, tc %1f \n", t, t_num, tc);
		tc = t*t_step;
		// printf("tc %1f \n", tc);

		//************************************
		// calculate derivatives 

		// calculate the photon observables
		// ap_a, a, a_a 
		calculate_photons<<<1,1>>>(tc,num_ens,para_a_dev,para_c_dev,\
					ap_a_dev,a_dev,a_a_dev,\
					a_sp_dev,sm_dev,a_sm_dev,\
					d_ap_a_dev,d_a_dev,d_a_a_dev);
		hipDeviceSynchronize();

		// calculate the atomic observables and atom-photon correlations
		// sz, sm, a_sz, a_sm, a_sp 
		calculate_atoms<<<1,num_ens>>>(tc,num_ens,para_a_dev,para_c_dev,\
						sz_dev,sm_dev,a_sz_dev,a_sm_dev,a_sp_dev,\
						sm_sp_dev,sm_sm_dev,sm_sz_dev,a_dev,ap_a_dev,a_a_dev,\
						d_sz_dev,d_sm_dev,d_a_sz_dev,d_a_sm_dev,d_a_sp_dev);
		hipDeviceSynchronize();

		// calculate the atom-atom correlations 
		// sm_sp, sm_sz, sm_sm, sz_sz
		calculate_correlations<<<num_ens,num_ens>>>(num_ens,t_step,para_a_dev,para_c_dev,\
							sm_sp_dev,sm_sz_dev,sm_sm_dev,sz_sz_dev,\
							a_dev,a_sm_dev,a_sp_dev,a_sz_dev,sm_dev,sz_dev,\
							d_sm_sp_dev,d_sm_sz_dev,d_sm_sm_dev,d_sz_sz_dev);
		hipDeviceSynchronize();

		//*************************************
		// update observables

		update_photons<<<1,1>>>(t_step,ap_a_dev,a_dev,a_a_dev,\
					d_ap_a_dev,d_a_dev,d_a_a_dev);
		hipDeviceSynchronize();


		update_atoms<<<1,num_ens>>>(num_ens,t_step,para_a_dev,sz_dev,sm_dev,a_sz_dev,a_sm_dev,a_sp_dev,\
					d_sz_dev,d_sm_dev,d_a_sz_dev,d_a_sm_dev,d_a_sp_dev);
		hipDeviceSynchronize();
		
		update_correlations<<<num_ens,num_ens>>>(num_ens,t_step,sm_sp_dev,sm_sz_dev,sm_sm_dev,sz_sz_dev,\
							d_sm_sp_dev,d_sm_sz_dev,d_sm_sm_dev,d_sz_sz_dev);
		hipDeviceSynchronize();

		if ( t%t_store == 0) {
		// copy the calculate observables back to CPU side 


			hipMemcpy(sz,sz_dev,num_ens*sizeof(double2),hipMemcpyDeviceToHost);
			hipMemcpy(sm,sm_dev,num_ens*sizeof(double2),hipMemcpyDeviceToHost);
			hipMemcpy(sm_sp,sm_sp_dev,num_ens*sizeof(double2),hipMemcpyDeviceToHost);
			hipMemcpy(&ap_a,ap_a_dev,sizeof(double2),hipMemcpyDeviceToHost);
			hipMemcpy(&a,a_dev,sizeof(double2),hipMemcpyDeviceToHost);
			hipDeviceSynchronize();

		// store the file
			fprintf(Result_time,"%e \n",(double)t*t_step);
			fprintf(Result_photon,"%e \n",ap_a.x);
			//printf("%1f	%e	\n",tc, ap_a.x);
		
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			for (int i = 0; i < num_ens; i++) {
				fprintf(Result_Sz,"%e ",sz[i].x);
				fprintf(Result_coherences_real,"%e ",sm_sp[i].x);
			}
			fprintf(Result_Sz,"\n");
			fprintf(Result_coherences_real,"\n");
			
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		}
	}


	// close the files
	fclose(Result_time);
	fclose(Result_Sz);
	fclose(Result_photon);
	fclose(Result_coherences_real);


	// close the memories 

	hipFree(para_a_dev); hipFree(para_c_dev); hipFree(t_step_dev);

	hipFree(ap_a_dev); hipFree(a_dev); hipFree(a_a_dev);

	hipFree(sz_dev); hipFree(sm_dev); hipFree(a_sz_dev);
	hipFree(a_sm_dev);hipFree(a_sp_dev);

	hipFree(sm_sp_dev); hipFree(sm_sz_dev);
	hipFree(sm_sm_dev); hipFree(sz_sz_dev);

	hipFree(d_ap_a_dev); hipFree(d_a_dev); hipFree(d_a_a_dev);

	hipFree(d_sz_dev); hipFree(d_sm_dev); hipFree(d_a_sz_dev);
	hipFree(d_a_sm_dev);hipFree(d_a_sp_dev);

	hipFree(d_sm_sp_dev); hipFree(d_sm_sz_dev);
	hipFree(d_sm_sm_dev); hipFree(d_sz_sz_dev);

	free(inhomo);
	free(N_a);
	free(omega_a);
	free(gamma_a);
	free(eta_a);
	free(chi_a);
	free(coup_a);
	free(loss_a);
	free(para_a);
	free(theta);
	free(phi);
	free(cu);
	free(cl);
	free(sz);
	free(sm);
	free(a_sz);
	free(a_sm);
	free(a_sp);
	free(sm_sp);
	free(sm_sz);
	free(sm_sm);
	free(sz_sz);

	end_clock = clock();
	// fprintf(stderr,"Program takes about %.2f s\n",(double)(ct1-ct0)/(double)CLOCKS_PER_SEC);
	printf("Program takes about %.2f s\n",(double)(end_clock - start_clock)/(double)CLOCKS_PER_SEC);
	return 0;
}

// Loads 2D matrix data from file (for e.g "Detuning.dat") 
// and assign it into given `out` 1D double array.
// 
// The size of `out` array MUST match the file or bigger.
// - size of `out` = number_of_rows * number_of_columns
// - if size of `out` is smaller, out-of-bound errors would occur.
// 
// The function expects the file to be in the following format:
// - Each row is separated by a newline '\n'.
// - Each column is separated by a tab '\t'.
// - Each row should not be more than 4096 characters.
int loadDoubleData(char* filename, double* out) {

	// File pointer to data file
	FILE* fp;

	// Open file 
	fp = fopen(filename, "r");
	if (fp == NULL) {
		printf("[loadDoubleData] Error: File (%s) does not exists\n", filename);
		// Return with error (1: file not found)
		return 1;
	}

    const char col_delim[] = "\t"; 	// Column delimiter
	char row[4096];		// This will store each fget attempt, might not be the entire line if the line exceeds buffer size
	char* col;			// This will store each tab-delimited value in the line
	int idx = 0;		// Keeps count of how many rows*columns we've parsed

	printf("[loadDoubleData] Reading data file: %s\n", filename);
	while (fgets(row, sizeof(row), fp)) {
		// TODO: check if the line read is complete (ends with newline)

		// Remove trailing newline
		row[strcspn(row, "\n")] = 0;

		// Split line by delimiter (tab)
		col = strtok(row, col_delim);
		while(col != NULL) {
			// Convert string to double value
			double val = strtod(col, NULL);

			// Add value to the out array
			out[idx++] = val;
			
			// Continue tokenizing the rest of the string
			col = strtok(NULL, col_delim);
		}
	}
	// Close file
	fclose(fp);
	
	// No error
	return 0;
}